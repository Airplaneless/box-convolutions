#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>
#include <THC/THC.h>

#define BLOCK_SIZE 256

using std::min;
using std::max;

#include "box_convolution.h" // for `enum class Parameter`

namespace gpu {

// TODO use constant memory when possible
// namespace constant {
//     __constant__ float xMinFrac[1536], xMaxFrac[1536];
//     __constant__ float yMinFrac[1536], yMaxFrac[1536];
//     __constant__ int xMinInt[1536], xMaxInt[1536];
//     __constant__ int yMinInt[1536], yMaxInt[1536];
//     __constant__ float area[1536];
// }

template <typename T, size_t N>
using CudaAcsr = const at::PackedTensorAccessor<T, N, at::RestrictPtrTraits, int32_t>;

// overload for "truncated"/"rounded" mode
template <bool normalize, typename scalar_t>
__global__ void boxConvUpdateOutputKernel(
    CudaAcsr<scalar_t,3> inputInt, CudaAcsr<scalar_t,3> output  ,
    CudaAcsr<int32_t ,1> xMinInt , CudaAcsr<int32_t ,1> xMaxInt ,
    CudaAcsr<int32_t ,1> yMinInt , CudaAcsr<int32_t ,1> yMaxInt ,
    CudaAcsr<scalar_t,1> area) {

    const int32_t y = blockDim.x * blockIdx.x + threadIdx.x;
    const int32_t x = blockDim.y * blockIdx.y + threadIdx.y;
    const int32_t inPlaneIdx = blockIdx.z / xMinInt.size(1);
    const int32_t paramIdx = blockIdx.z % (xMinInt.size(0) * xMinInt.size(1));
    const int32_t h = output.size(1);
    const int32_t w = output.size(2);

    if (x < h and y < w) {
        // Must add 1 to xMax/yMax/xMin/yMin due to OpenCV's
        // `integral()` behavior. Namely, I(x,0) and I(0,y) are
        // always 0 (so it's a C-style array sum).

        // However, when computing sums, we subtract values at points 
        // like y+yMin-1 and x+xMin-1, so we also SUBTRACT 1 from xMin
        // and yMin, and thus finally they are not affected.

        const int32_t t = max(0, min(x+xMinInt[paramIdx], h));
        const int32_t b = max(0, min(x+xMaxInt[paramIdx], h));
        const int32_t l = max(0, min(y+yMinInt[paramIdx], w));
        const int32_t r = max(0, min(y+yMaxInt[paramIdx], w));

        scalar_t outValue = 0;

        outValue += inputInt[inPlaneIdx][b][r];
        outValue -= inputInt[inPlaneIdx][t][r];
        outValue -= inputInt[inPlaneIdx][b][l];
        outValue += inputInt[inPlaneIdx][t][l];

        // TODO error: expression must be a modifiable lvalue
        output.data()[(blockIdx.z * h + x) * w + y] =
            outValue * (normalize ? area[paramIdx] : static_cast<scalar_t>(1));
    }
}

// overload for "exact" mode
template <bool normalize, typename scalar_t>
__global__ void boxConvUpdateOutputKernel(
    CudaAcsr<scalar_t,3> inputInt, CudaAcsr<scalar_t,3> output  ,
    CudaAcsr<int32_t ,1> xMinInt , CudaAcsr<int32_t ,1> xMaxInt ,
    CudaAcsr<int32_t ,1> yMinInt , CudaAcsr<int32_t ,1> yMaxInt ,
    CudaAcsr<scalar_t,1> xMinFrac, CudaAcsr<scalar_t,1> xMaxFrac,
    CudaAcsr<scalar_t,1> yMinFrac, CudaAcsr<scalar_t,1> yMaxFrac,
    CudaAcsr<scalar_t,1> area) {

    const int32_t y = blockDim.x * blockIdx.x + threadIdx.x;
    const int32_t x = blockDim.y * blockIdx.y + threadIdx.y;
    const int32_t inPlaneIdx = blockIdx.z / xMinInt.size(1);
    const int32_t paramIdx = blockIdx.z % (xMinInt.size(0) * xMinInt.size(1));
    const int32_t h = output.size(1);
    const int32_t w = output.size(2);

    if (x < h and y < w) {
        // Must add 1 to xMax/yMax/xMin/yMin due to OpenCV's
        // `integral()` behavior. Namely, I(x,0) and I(0,y) are
        // always 0 (so it's a C-style array sum).

        // However, when computing sums, we subtract values at points 
        // like y+yMin-1 and x+xMin-1, so we also SUBTRACT 1 from xMin
        // and yMin, and thus finally they are not affected.
        const int xMinCurr = xMinInt[paramIdx];
        const int xMaxCurr = xMaxInt[paramIdx];
        const int yMinCurr = yMinInt[paramIdx];
        const int yMaxCurr = yMaxInt[paramIdx];

        const scalar_t xMinCurrFrac = xMinFrac[paramIdx];
        const scalar_t xMaxCurrFrac = xMaxFrac[paramIdx];
        const scalar_t yMinCurrFrac = yMinFrac[paramIdx];
        const scalar_t yMaxCurrFrac = yMaxFrac[paramIdx];

        const int32_t t = max(0, min(x+xMinCurr, h));
        const int32_t b = max(0, min(x+xMaxCurr, h));
        const int32_t l = max(0, min(y+yMinCurr, w));
        const int32_t r = max(0, min(y+yMaxCurr, w));

        const int32_t bAdv = max(0, min(x+xMaxCurr+1, h));
        const int32_t rAdv = max(0, min(y+yMaxCurr+1, w));
        const int32_t tAdv = max(0, min(x+xMinCurr-1, h));
        const int32_t lAdv = max(0, min(y+yMinCurr-1, w));

        scalar_t outValue;

        // -- main area
        outValue = 
              inputInt[inPlaneIdx][b][r]
            - inputInt[inPlaneIdx][t][r]
            - inputInt[inPlaneIdx][b][l]
            + inputInt[inPlaneIdx][t][l];

        // -- xMax border
        outValue +=
            ( inputInt[inPlaneIdx][bAdv][r]
            - inputInt[inPlaneIdx][b   ][r]
            - inputInt[inPlaneIdx][bAdv][l]
            + inputInt[inPlaneIdx][b   ][l]) * xMaxCurrFrac;

        // -- yMax border
        outValue +=
            ( inputInt[inPlaneIdx][b][rAdv]
            - inputInt[inPlaneIdx][b][r   ]
            - inputInt[inPlaneIdx][t][rAdv]
            + inputInt[inPlaneIdx][t][r   ]) * yMaxCurrFrac;

        // -- xMin border
        outValue +=
            ( inputInt[inPlaneIdx][t   ][r]
            - inputInt[inPlaneIdx][tAdv][r]
            - inputInt[inPlaneIdx][t   ][l]
            + inputInt[inPlaneIdx][tAdv][l]) * xMinCurrFrac;

        // -- yMin border
        outValue +=
            ( inputInt[inPlaneIdx][b][l   ]
            - inputInt[inPlaneIdx][b][lAdv]
            - inputInt[inPlaneIdx][t][l   ]
            + inputInt[inPlaneIdx][t][lAdv]) * yMinCurrFrac;

        // -- corner pixels
        // Note: before, I used plain `input` to access corner values
        // with lower memory access overhead. Moved to `input_integrated`
        // to get rid of an extra input to this function.
        if (not ((x+xMaxCurr >= h) | (y+yMaxCurr >= w) |
                 (x+xMaxCurr <  0) | (y+yMaxCurr <  0))) {
            outValue += 
                xMaxCurrFrac * yMaxCurrFrac *
                ( inputInt[inPlaneIdx][b+1][r+1]
                - inputInt[inPlaneIdx][b  ][r+1]
                - inputInt[inPlaneIdx][b+1][r  ]
                + inputInt[inPlaneIdx][b  ][r  ]);
        }

        if (not ((x+xMinCurr >  h) | (y+yMaxCurr >= w) |
                 (x+xMinCurr <= 0) | (y+yMaxCurr <  0))) {
            outValue +=
                xMinCurrFrac * yMaxCurrFrac *
                ( inputInt[inPlaneIdx][t  ][r+1]
                - inputInt[inPlaneIdx][t-1][r+1]
                - inputInt[inPlaneIdx][t  ][r  ]
                + inputInt[inPlaneIdx][t-1][r  ]);
        }

        if (not ((x+xMaxCurr >= h) | (y+yMinCurr >  w) |
                 (x+xMaxCurr <  0) | (y+yMinCurr <= 0))) {
            outValue +=
                xMaxCurrFrac * yMinCurrFrac *
                ( inputInt[inPlaneIdx][b+1][l  ]
                - inputInt[inPlaneIdx][b  ][l  ]
                - inputInt[inPlaneIdx][b+1][l-1]
                + inputInt[inPlaneIdx][b  ][l-1]);
        }

        if (not ((x+xMinCurr >  h) | (y+yMinCurr >  w) |
                 (x+xMinCurr <= 0) | (y+yMinCurr <= 0))) {
            outValue +=
                xMinCurrFrac * yMinCurrFrac *
                ( inputInt[inPlaneIdx][t  ][l  ]
                - inputInt[inPlaneIdx][t-1][l  ]
                - inputInt[inPlaneIdx][t  ][l-1]
                + inputInt[inPlaneIdx][t-1][l-1]);
        }

        // TODO error: expression must be a modifiable lvalue
        output.data()[(blockIdx.z * h + x) * w + y] =
            outValue * (normalize ? area[paramIdx] : static_cast<scalar_t>(1));
    }
}

// TODO put split params and area into constant memory
template <bool normalize, bool exact>
void boxConvUpdateOutput(
    at::Tensor & xMinInt , at::Tensor & xMaxInt , at::Tensor & yMinInt , at::Tensor & yMaxInt ,
    at::Tensor & xMinFrac, at::Tensor & xMaxFrac, at::Tensor & yMinFrac, at::Tensor & yMaxFrac,
    at::Tensor & area, at::Tensor & input_integrated, at::Tensor & output) {

    const int h = output.size(-2);
    const int w = output.size(-1);
    const int totalOutputChannels = output.numel() / (h * w);

    const dim3 blockSize(32, 32, 1);
    const dim3 gridSize(
        (w + blockSize.x - 1) / blockSize.x,
        (h + blockSize.y - 1) / blockSize.y,
        (totalOutputChannels  + blockSize.z - 1) / blockSize.z);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(output.type(), "gpu::boxConvUpdateOutput", ([&] {

        auto xMinIntFlat = xMinInt.view(-1);
        auto xMaxIntFlat = xMaxInt.view(-1);
        auto yMinIntFlat = yMinInt.view(-1);
        auto yMaxIntFlat = yMaxInt.view(-1);
        auto xMinIntAcsr = xMinIntFlat.packed_accessor<int32_t, 1, at::RestrictPtrTraits, int32_t>();
        auto xMaxIntAcsr = xMaxIntFlat.packed_accessor<int32_t, 1, at::RestrictPtrTraits, int32_t>();
        auto yMinIntAcsr = yMinIntFlat.packed_accessor<int32_t, 1, at::RestrictPtrTraits, int32_t>();
        auto yMaxIntAcsr = yMaxIntFlat.packed_accessor<int32_t, 1, at::RestrictPtrTraits, int32_t>();

        auto xMinFracFlat = xMinFrac.view(-1);
        auto xMaxFracFlat = xMaxFrac.view(-1);
        auto yMinFracFlat = yMinFrac.view(-1);
        auto yMaxFracFlat = yMaxFrac.view(-1);
        auto xMinFracAcsr = xMinFracFlat.packed_accessor<scalar_t, 1, at::RestrictPtrTraits, int32_t>();
        auto xMaxFracAcsr = xMaxFracFlat.packed_accessor<scalar_t, 1, at::RestrictPtrTraits, int32_t>();
        auto yMinFracAcsr = yMinFracFlat.packed_accessor<scalar_t, 1, at::RestrictPtrTraits, int32_t>();
        auto yMaxFracAcsr = yMaxFracFlat.packed_accessor<scalar_t, 1, at::RestrictPtrTraits, int32_t>();
        
        auto areaAcsr = xMinFracAcsr; // because there's no default ctor :(
        // only initialize the accessor if `area` is defined (errors otherwise)
        if (normalize) {
            auto areaFlat = area.view(-1);
            areaAcsr = areaFlat.packed_accessor<scalar_t, 1, at::RestrictPtrTraits, int32_t>();
        }

        auto inputIntFlattened = input_integrated.view({-1, h+1, w+1});
        auto inputIntAcsr =
            inputIntFlattened.packed_accessor<scalar_t, 3, at::RestrictPtrTraits, int32_t>();
            
        auto outputFlattened = output.view({-1, h, w});
        auto outputAcsr = 
            outputFlattened.packed_accessor<scalar_t, 3, at::RestrictPtrTraits, int32_t>();

        if (exact) {
            boxConvUpdateOutputKernel <normalize>
                <<<gridSize, blockSize, 0, at::cuda::getCurrentCUDAStream()>>> (
                inputIntAcsr, outputAcsr,
                xMinIntAcsr , xMaxIntAcsr , yMinIntAcsr , yMaxIntAcsr,
                xMinFracAcsr, xMaxFracAcsr, yMinFracAcsr, yMaxFracAcsr,
                areaAcsr);
        } else {
            boxConvUpdateOutputKernel <normalize>
                <<<gridSize, blockSize, 0, at::cuda::getCurrentCUDAStream()>>> (
                inputIntAcsr, outputAcsr,
                xMinIntAcsr, xMaxIntAcsr, yMinIntAcsr, yMaxIntAcsr,
                areaAcsr);
        }
        THCudaCheck(hipGetLastError());
    }));
}

// explicitly instantiate
template void boxConvUpdateOutput<true, true>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &);

template void boxConvUpdateOutput<false, true>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &);

template void boxConvUpdateOutput<true, false>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &);

template void boxConvUpdateOutput<false, false>(
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &, at::Tensor &,
    at::Tensor &, at::Tensor &, at::Tensor &);

}
